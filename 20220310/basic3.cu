#include <stdio.h>

/*
 *
 * struct hipDeviceProp_t {
 *     char name[256]; // device name
 *     size_t totoalGlobalMem;
 *     size_t sharedMemPerBlock;
 *     int regsPerBlock;
 *     int wrapSize;
 *     size_t memPitch;
 *     int maxThreadsPerBlock;
 *     int maxThreadsDim[3];
 *     int maxGridSize[3];
 *     size_t totalConstMem;
 *     int major;     // device major number
 *     int minor;     // device minor number
 *     int clockRate;
 *     size_t textureAlignment;
 *     int deviceOverlap;
 *     int multiProcessorCount;
 *     int kernelExecTimeoutEnabled;
 *     int integrated;
 *     int canMapHostMemory;
 *     int computeMode;
 *     int maxTexture1D;
 *     int maxTexture2D[2];
 *     int maxTexture3D[3];
 *     int maxTexture3dArray[3];
 *     int concurrentKernels;
 * };
 */

int main(void){
    hipDeviceProp_t prop;
    int count;

    hipGetDeviceCount(&count);
    for(int i = 0; i < count; i++){
        hipGetDeviceProperties(&prop, i);
        printf("--------- Dveice %d ---------\n", i);
        printf("Name: %s\n", prop.name);
        printf("Capability: %d.%d\n", prop.major, prop.minor);
        printf("Clock Rate: %d\n", prop.clockRate);
        printf("Device copy overlap: ");
        if(prop.deviceOverlap)
            printf("Enabled!\n");
        else
            printf("Disabled!\n");
        printf("-----------------------------\n");
    }

    return 0;
}