
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 100

__global__ void add(int* a){
    int tid = blockIdx.x;
    for(int i = 0; i <= tid; i++)
        a[tid] += i;
    printf("Calculate Complete: %d\n", tid);
}

int main(void){
    int a[N] = {0,};
    int *dev_a;      // 'a' array value for device 0
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice); // copy 'a' array value
    add<<<N, 1>>>(dev_a);
    hipMemcpy(a, dev_a, N * sizeof(int), hipMemcpyDeviceToHost); 
    printf("Result: ");
    for(int i = 0; i < N; i++){
        printf("%d ", a[i]);
    }
    hipFree(dev_a);
    printf("\n");
    return 0;
}