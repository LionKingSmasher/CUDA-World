
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int n1, int n2, int* n3) {
    printf("Calculate!\n");
    *n3 = n1 + n2;
}

int main(void){
    int c = 0;
    int *p_c;
    hipMalloc((void**)&p_c, sizeof(int));
    add<<<1,1>>>(1, 1, p_c);
    hipMemcpy(&c, p_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("1 + 1 = %d\n", c);
    hipFree(p_c);
    return 0;
}